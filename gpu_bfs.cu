#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <sys/time.h>

using namespace std;

#include "bfs_kernel.cu"

const int MAX_THREADS_PER_BLOCK = 256;
char *infile = NULL;

void usage(char* prog_name, const char* more)
{
	cerr << more;
	cerr << "usage: " << prog_name << "input_file " << endl;
	exit(0);
}

void parse_args(int argc, char** argv) 
{
	for (int i = 0; i < argc; i++)
	{
		if (argv[i][0] == '-')
		{
			switch(argv[i][1])
			{
				case 'i':
					if (i == argc - 1) {
						usage(argv[0], "Infile missing");
					}
					infile = argv[i + 1];
					i++;
					break;
			}
		}
	}
}

void bfsGraph(char *filename, int start_position, char *outFile)
{
	int nb_nodes;
	ifstream finput;
	finput.open(filename, ios::in | ios::binary);

	// Read number of nodes, first 4 bytes of file
	finput.read((char*)&nb_nodes, 4);

	if (start_position < 0 || start_position > nb_nodes)
		return;

	// Read cumulative degrees, 4 bytes per node
	int *degrees = new int[nb_nodes];
	finput.read((char*) degrees, nb_nodes * 4);
	int *starting = new int[nb_nodes];
	memset(starting, 0, sizeof(int) * nb_nodes);
	for (int i = 1; i < nb_nodes; i++)
	{
		starting[i] = degrees[i - 1];
	}

	// Read links, 4 bytes per link
	int nb_links = degrees[nb_nodes - 1];
	int *links = new int[nb_links];
	finput.read((char*) links, nb_links * 4);
	finput.close();

	//cout << "Number of nodes: " << nb_nodes << endl;
	//cout << "Number of links: " << nb_links << endl;

	// Determine number of blocks and threads
	int num_of_blocks = 1;
	int num_of_threads_per_block = nb_nodes;
	if (nb_nodes > MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int) ceil((double) nb_nodes/
			(double) MAX_THREADS_PER_BLOCK);
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
	}

	// allocate host memory
	Node *h_graph_nodes = (Node *) malloc(sizeof(Node) * nb_nodes);
	bool* h_graph_level = (bool *) malloc(sizeof(bool) * nb_nodes);
	bool* h_graph_visited = (bool *) malloc(sizeof(bool) * nb_nodes);

	// Initialize memory of nodes
	h_graph_nodes[0].starting = 0;
	h_graph_nodes[0].no_of_edges = degrees[0];
	h_graph_level[0] = false;
	h_graph_visited[0] = false;
	for (unsigned int i = 1; i < nb_nodes; i++)
	{
		h_graph_nodes[i].starting = starting[i];
		h_graph_nodes[i].no_of_edges = degrees[i] - degrees[i-1];
		h_graph_level[i] = false;
		h_graph_visited[i] = false;
	}
	h_graph_level[start_position] = true;
	h_graph_visited[start_position] = true;

	// Copy node list to cuda memory
	Node *d_graph_nodes;
	hipMalloc((void **) &d_graph_nodes, sizeof(Node) * nb_nodes);
	hipMemcpy(d_graph_nodes, h_graph_nodes, sizeof(Node) *
		nb_nodes, hipMemcpyHostToDevice);

	// Copy edge list to device memory
	int *d_edge_list;
	hipMalloc((void **) &d_edge_list, sizeof(int) * nb_links);
	hipMemcpy(d_edge_list, links, sizeof(int) * nb_links,
		hipMemcpyHostToDevice);

	// Copy the visted array to device memory
	bool *d_graph_visited;
	hipMalloc((void **) &d_graph_visited, sizeof(bool) * nb_nodes);
	hipMemcpy(d_graph_visited, h_graph_visited, sizeof(bool) *
		nb_nodes, hipMemcpyHostToDevice);

	// Copy the level to device memory
	bool* d_graph_level;
	hipMalloc((void **) &d_graph_level, sizeof(bool) * nb_nodes);
	hipMemcpy(d_graph_level, h_graph_level, sizeof(bool) * nb_nodes,
		hipMemcpyHostToDevice);

	// Allocate memory for the result on host
	int *h_cost = (int *) malloc(sizeof(int) * nb_nodes);
	for (int i = 0; i < nb_nodes; i++)
	{
		h_cost[i] = -1;
	}
	h_cost[start_position] = 0;

	// Allocate device memory for result
	int *d_cost;
	hipMalloc((void **) &d_cost, sizeof(int) * nb_nodes);
	hipMemcpy(d_cost, h_cost, sizeof(int) * nb_nodes,
		hipMemcpyHostToDevice);

	bool *d_over;
	hipMalloc((void **) &d_over, sizeof(bool));
	bool stop;

	struct timeval start, end;    
	gettimeofday(&start, NULL);
	do 
	{
		stop = false;
		hipMemcpy(d_over, &stop, sizeof(bool), 
			hipMemcpyHostToDevice);
		bfs_kernel<<<num_of_blocks, 
		num_of_threads_per_block>>>(d_graph_nodes, d_edge_list,
			d_graph_level, d_graph_visited, d_cost, d_over,
			nb_nodes);
		hipDeviceSynchronize();

		hipMemcpy(&stop, d_over, sizeof(bool),
			hipMemcpyDeviceToHost);
		//cout << "stop : " << stop << endl;
		stop = false;
	} while(stop);

	gettimeofday(&end, NULL);
	printf("%ld\n",
           (end.tv_sec * 1000000 + end.tv_usec)
           - (start.tv_sec * 1000000 + start.tv_usec));
	
	hipMemcpy(h_cost, d_cost, sizeof(int) * nb_nodes,
		hipMemcpyDeviceToHost);
	
	hipMemcpy(h_graph_visited, d_graph_visited, sizeof(bool) *
			nb_nodes, hipMemcpyDeviceToHost);
	
	//cout << "15152: " << h_graph_visited[15152] << " " << h_cost[15152] << endl;

	//cout << "success!" << endl;

	// sanity check against the serial BFS code	
	/*
	cout << degrees[0] << endl;
	for (int i = 0; i < 16; i++)
	{
		cout << "(" << links[i] << ")" << endl;
	}

	FILE *fpt = fopen("1092-links.txt", "w");
	for (int i = starting[1092]; i < degrees[1092]; i++)
	{
		fprintf(fpt, "(%d)\n", links[i]);
	}
	fclose(fpt);
	*/

	
	

	// Store results into a file
	FILE *fpo = fopen(outFile, "w");
	for (int i = 0; i < nb_nodes; i++)
	{
		fprintf(fpo, "(%d) cost:%d\n", i, h_cost[i]);
	}
	fclose(fpo);
	
	// clean up memory
	free(h_graph_nodes);
	free(links);
	free(h_graph_level);
	free(h_graph_visited);
	free(h_cost);
	hipFree(d_graph_nodes);
	hipFree(d_edge_list);
	hipFree(d_graph_level);
	hipFree(d_graph_visited);
	hipFree(d_cost);

}

int main(int argc, char **argv)
{
	//parse_args(argc, argv);
	char *filename = argv[1];
	char*outFile = argv[2];
	bfsGraph(filename, 0, outFile);
	return 0;
}